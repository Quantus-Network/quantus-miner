/**
 * Quantus External Miner - CUDA Kernel (G1 bring-up)
 *
 * This kernel provides a minimal, correctness-first pipeline for the GPU path:
 * - Implement 512-bit Montgomery multiplication (8×64-bit limbs) on device.
 * - Keep y in Montgomery domain during iteration and convert to normal domain before output.
 * - For bring-up (G1), the kernel writes normalized y values back to host memory.
 *   The host will compute SHA3-512(y_be64) and distances for parity validation.
 *
 * Notes:
 * - Limbs are little-endian: limb 0 is the least significant 64 bits.
 * - CIOS Montgomery reduction is used with 64×64→128 products via __umul64hi.
 * - This skeleton intentionally excludes early-exit and on-device SHA3; those are part of G2.
 *
 * Build:
 * - The engine-gpu-cuda crate provides a build.rs that compiles this .cu into PTX when the
 *   "cuda" feature is enabled, placing artifacts under $OUT_DIR and exposing ENGINE_GPU_CUDA_PTX_DIR.
 */

#include <stdint.h>
#include <hip/hip_runtime.h>

extern "C" {

// -------------------------------------------------------------------------------------------------
// Utilities: 64×64→128 multiply (lo, hi), add-with-carry helpers, compare/subtract
// -------------------------------------------------------------------------------------------------

__device__ __forceinline__ void mul64wide(uint64_t a, uint64_t b, uint64_t &lo, uint64_t &hi) {
    lo = a * b;
    hi = __umul64hi(a, b);
}

// sum := x + y, carry_out returns 0 or 1
__device__ __forceinline__ uint64_t add64_carry(uint64_t x, uint64_t y, uint64_t &carry_out) {
    uint64_t s = x + y;
    carry_out = (s < x) ? 1ull : 0ull;
    return s;
}

// sum := x + y + carry_in, carry_out returns 0 or 1
__device__ __forceinline__ uint64_t add64_2carry(uint64_t x, uint64_t y, uint64_t carry_in, uint64_t &carry_out) {
    uint64_t s1 = x + y;
    uint64_t c1 = (s1 < x) ? 1ull : 0ull;
    uint64_t s2 = s1 + carry_in;
    uint64_t c2 = (s2 < s1) ? 1ull : 0ull;
    carry_out = c1 + c2;
    return s2;
}

// return true if a (LE limbs) >= b (LE limbs), by numeric value
__device__ __forceinline__ bool ge_le_8(const uint64_t a[8], const uint64_t b[8]) {
    // Compare from most significant limb to least
    for (int i = 7; i >= 0; --i) {
        if (a[i] != b[i]) {
            return a[i] > b[i];
        }
    }
    return true; // equal
}

// a := a - b (LE limbs)
__device__ __forceinline__ void sub_le_in_place_8(uint64_t a[8], const uint64_t b[8]) {
    uint64_t borrow = 0;
    for (int i = 0; i < 8; ++i) {
        uint64_t bi = b[i];
        uint64_t ai = a[i];
        uint64_t tmp = ai - bi - borrow;
        // borrow occurs if ai < (bi + borrow)
        uint64_t needed = (ai < bi) || (borrow && ai == bi) ? 1ull : 0ull;
        a[i] = tmp;
        borrow = needed;
    }
}

// Convert 8 LE limbs into 64 BE bytes into out[64] (for host hashing later, if needed)
// Not used inside the kernel (G1 writes limbs), but kept here for reference.
// __device__ __forceinline__ void le8_to_be64(const uint64_t le[8], uint8_t out[64]) {
//     for (int i = 0; i < 8; ++i) {
//         uint64_t limb = le[7 - i]; // most significant limb first
//         for (int b = 0; b < 8; ++b) {
//             out[i * 8 + (7 - b)] = (uint8_t)((limb >> (b * 8)) & 0xFF);
//         }
//     }
// }

// -------------------------------------------------------------------------------------------------
// Montgomery arithmetic (CIOS) for 512-bit numbers (8×64-bit limbs), little-endian.
// -------------------------------------------------------------------------------------------------

// out <- (a * b * R^{-1}) mod n
__device__ __forceinline__ void mont_mul_512(
    const uint64_t a[8],
    const uint64_t b[8],
    const uint64_t n[8],
    const uint64_t n0_inv,
    uint64_t out[8]
) {
    // 9-limb accumulator (LE); accumulates 128-bit intermediates via split-add with carries
    uint64_t acc[9];
#pragma unroll
    for (int k = 0; k < 9; ++k) acc[k] = 0ull;

    for (int i = 0; i < 8; ++i) {
        // acc += a[i] * b
        uint64_t ai = a[i];
        uint64_t carry = 0ull;
#pragma unroll
        for (int j = 0; j < 8; ++j) {
            uint64_t lo, hi;
            mul64wide(ai, b[j], lo, hi);

            // acc[j] += lo + carry, propagate carry to hi
            uint64_t c0, c1;
            uint64_t s0 = add64_carry(acc[j], lo, c0);
            uint64_t s1 = add64_carry(s0, carry, c1);
            acc[j] = s1;
            // new carry = hi + c0 + c1
            carry = hi + c0 + c1;
        }
        // acc[8] += carry
        uint64_t c_acc8;
        acc[8] = add64_carry(acc[8], carry, c_acc8);
        // c_acc8 overflow beyond 9th limb is discarded (by design in CIOS with next steps)

        // m = (acc[0] * n0_inv) mod 2^64
        uint64_t m = (uint64_t)(acc[0] * n0_inv);

        // acc += m * n
        uint64_t carry2 = 0ull;
#pragma unroll
        for (int j = 0; j < 8; ++j) {
            uint64_t lo2, hi2;
            mul64wide(m, n[j], lo2, hi2);

            uint64_t c0, c1;
            uint64_t s0 = add64_carry(acc[j], lo2, c0);
            uint64_t s1 = add64_carry(s0, carry2, c1);
            acc[j] = s1;
            carry2 = hi2 + c0 + c1;
        }
        // acc[8] += carry2
        uint64_t c_acc8_b;
        acc[8] = add64_carry(acc[8], carry2, c_acc8_b);

        // Shift acc right by one limb (drop acc[0])
#pragma unroll
        for (int j = 0; j < 8; ++j) {
            acc[j] = acc[j + 1];
        }
        acc[8] = 0ull;
    }

    // Conditional subtract: if acc >= n, subtract n
    if (ge_le_8(acc, n)) {
        sub_le_in_place_8(acc, n);
    }

#pragma unroll
    for (int i = 0; i < 8; ++i) {
        out[i] = acc[i];
    }
}

// to_mont(x) = x * R^2 mod n
__device__ __forceinline__ void to_mont_512(
    const uint64_t x[8],
    const uint64_t r2[8],
    const uint64_t n[8],
    const uint64_t n0_inv,
    uint64_t out[8]
) {
    mont_mul_512(x, r2, n, n0_inv, out);
}

// from_mont(x̂) = x̂ * 1 mod n
__device__ __forceinline__ void from_mont_512(
    const uint64_t xhat[8],
    const uint64_t n[8],
    const uint64_t n0_inv,
    uint64_t out[8]
) {
    // Multiply by 1 (Montgomery): one = [1,0,..,0]
    uint64_t one[8];
#pragma unroll
    for (int i = 0; i < 8; ++i) one[i] = 0ull;
    one[0] = 1ull;
    mont_mul_512(xhat, one, n, n0_inv, out);
}

// -------------------------------------------------------------------------------------------------
// Kernel: G1 bring-up
//
// Each thread:
//  - Loads y0 (normal domain) for that thread.
//  - Computes y_hat0 = to_mont(y0) on device.
//  - Iterates iters_per_thread times:
//      y_hat = mont_mul(y_hat, m_hat)
//      y = from_mont(y_hat)
//      Writes y (LE limbs) to y_out at [thread_offset + iter]
// -------------------------------------------------------------------------------------------------

extern "C" __global__ void qpow_montgomery_g1_kernel(
    // Per-job constants (each 8 limbs, LE)
    const uint64_t* __restrict__ m,        // not used in G1 directly (we pass m_hat)
    const uint64_t* __restrict__ n,
    const uint64_t  n0_inv,
    const uint64_t* __restrict__ r2,
    const uint64_t* __restrict__ m_hat,

    // Per-thread starting state (normal domain)
    const uint64_t* __restrict__ y0,       // length: num_threads * 8 limbs

    // Output buffer for normalized y (for host SHA3 in G1)
    uint64_t* __restrict__ y_out,          // length: num_threads * iters_per_thread * 8 limbs

    // Threading parameters
    const uint32_t num_threads,
    const uint32_t iters_per_thread
) {
    const uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_threads) {
        return;
    }

    // Local copies of constants (consider placing in __constant__ memory for G2+)
    uint64_t n_loc[8], r2_loc[8], mhat_loc[8];
#pragma unroll
    for (int i = 0; i < 8; ++i) {
        n_loc[i]    = n[i];
        r2_loc[i]   = r2[i];
        mhat_loc[i] = m_hat[i];
    }

    // Load this thread's y0 (normal domain)
    uint64_t y0_loc[8];
#pragma unroll
    for (int i = 0; i < 8; ++i) {
        y0_loc[i] = y0[tid * 8u + i];
    }

    // Transform to Montgomery domain
    uint64_t yhat[8];
    to_mont_512(y0_loc, r2_loc, n_loc, n0_inv, yhat);

    // Iterate and emit normalized y per step
    // Output stride per thread: iters_per_thread * 8 limbs
    uint64_t* out_base = y_out + (static_cast<size_t>(tid) * static_cast<size_t>(iters_per_thread) * 8ull);

    for (uint32_t iter = 0; iter < iters_per_thread; ++iter) {
        // y_hat = y_hat * m_hat
        uint64_t yhat_next[8];
        mont_mul_512(yhat, mhat_loc, n_loc, n0_inv, yhat_next);

#pragma unroll
        for (int i = 0; i < 8; ++i) {
            yhat[i] = yhat_next[i];
        }

        // y = from_mont(y_hat)
        uint64_t y_norm[8];
        from_mont_512(yhat, n_loc, n0_inv, y_norm);

        // Store normalized y (LE limbs) for host SHA3 and distance validation
#pragma unroll
        for (int i = 0; i < 8; ++i) {
            out_base[iter * 8u + i] = y_norm[i];
        }
    }
}

// -------------------------------------------------------------------------------------------------
// Host-callable launcher wrapper (optional; typically loaded via PTX and launched from Rust)
// -------------------------------------------------------------------------------------------------

// The Rust host side will load this kernel from PTX and launch it using `cust`/`rustacuda`.
// Example signature in Rust (pseudo):
//
// launch!(module.qpow_montgomery_g1_kernel<<<grid, block, 0, stream>>>(
//     d_m.as_device_ptr(),
//     d_n.as_device_ptr(),
//     n0_inv,
//     d_r2.as_device_ptr(),
//     d_mhat.as_device_ptr(),
//     d_y0.as_device_ptr(),
//     d_y_out.as_device_ptr(),
//     num_threads,
//     iters_per_thread
// ))?;
//
// Note: For G1, the host will compute SHA3-512(y) and distances on the CPU,
// validating correctness against cpu-fast/cpu-montgomery on small ranges.
//
 
// -------------------------------------------------------------------------------------------------
// G2 additions: device SHA3-512, threshold compare, and early-exit
// -------------------------------------------------------------------------------------------------
 
// 64-bit rotate-left
__device__ __forceinline__ uint64_t rotl64(uint64_t x, int n) {
    return (x << n) | (x >> (64 - n));
}
 
// Keccak-f[1600] round constants
__device__ __constant__ uint64_t KECCAK_RC[24] = {
    0x0000000000000001ULL, 0x0000000000008082ULL,
    0x800000000000808aULL, 0x8000000080008000ULL,
    0x000000000000808bULL, 0x0000000080000001ULL,
    0x8000000080008081ULL, 0x8000000000008009ULL,
    0x000000000000008aULL, 0x0000000000000088ULL,
    0x0000000080008009ULL, 0x000000008000000aULL,
    0x000000008000808bULL, 0x800000000000008bULL,
    0x8000000000008089ULL, 0x8000000000008003ULL,
    0x8000000000008002ULL, 0x8000000000000080ULL,
    0x000000000000800aULL, 0x800000008000000aULL,
    0x8000000080008081ULL, 0x8000000000008080ULL,
    0x0000000080000001ULL, 0x8000000080008008ULL
};
 
// Load/store helpers (little-endian)
__device__ __forceinline__ uint64_t load64_le(const uint8_t* p) {
    return ((uint64_t)p[0])       |
           ((uint64_t)p[1] << 8)  |
           ((uint64_t)p[2] << 16) |
           ((uint64_t)p[3] << 24) |
           ((uint64_t)p[4] << 32) |
           ((uint64_t)p[5] << 40) |
           ((uint64_t)p[6] << 48) |
           ((uint64_t)p[7] << 56);
}
__device__ __forceinline__ void store64_le(uint8_t* p, uint64_t v) {
    p[0] = (uint8_t)(v);
    p[1] = (uint8_t)(v >> 8);
    p[2] = (uint8_t)(v >> 16);
    p[3] = (uint8_t)(v >> 24);
    p[4] = (uint8_t)(v >> 32);
    p[5] = (uint8_t)(v >> 40);
    p[6] = (uint8_t)(v >> 48);
    p[7] = (uint8_t)(v >> 56);
}
 
// Keccak-f[1600] permutation
__device__ __forceinline__ void keccak_f1600(uint64_t s[25]) {
#pragma unroll
    for (int round = 0; round < 24; ++round) {
        // Theta
        uint64_t C[5], D[5];
#pragma unroll
        for (int x = 0; x < 5; ++x) {
            C[x] = s[x] ^ s[x + 5] ^ s[x + 10] ^ s[x + 15] ^ s[x + 20];
        }
#pragma unroll
        for (int x = 0; x < 5; ++x) {
            D[x] = C[(x + 4) % 5] ^ rotl64(C[(x + 1) % 5], 1);
        }
#pragma unroll
        for (int x = 0; x < 5; ++x) {
#pragma unroll
            for (int y = 0; y < 5; ++y) {
                s[x + 5 * y] ^= D[x];
            }
        }
 
        // Rho and Pi
        uint64_t B[25];
        const int r[25] = {
            0,  1, 62, 28, 27,
           36, 44,  6, 55, 20,
            3, 10, 43, 25, 39,
           41, 45, 15, 21,  8,
           18,  2, 61, 56, 14
        };
#pragma unroll
        for (int x = 0; x < 5; ++x) {
#pragma unroll
            for (int y = 0; y < 5; ++y) {
                int idx = x + 5 * y;
                int X = y;
                int Y = (2 * x + 3 * y) % 5;
                B[X + 5 * Y] = rotl64(s[idx], r[idx]);
            }
        }
 
        // Chi
#pragma unroll
        for (int x = 0; x < 5; ++x) {
#pragma unroll
            for (int y = 0; y < 5; ++y) {
                s[x + 5 * y] = B[x + 5 * y] ^ ((~B[((x + 1) % 5) + 5 * y]) & B[((x + 2) % 5) + 5 * y]);
            }
        }
 
        // Iota
        s[0] ^= KECCAK_RC[round];
    }
}
 
// Device SHA3-512 for a single 64-byte message
__device__ __forceinline__ void sha3_512_64bytes(const uint8_t in_be64[64], uint8_t out_be64[64]) {
    // Initialize state to zero
    uint64_t s[25];
#pragma unroll
    for (int i = 0; i < 25; ++i) s[i] = 0ull;
 
    // Absorb (rate = 72 bytes). Message is 64 bytes: append 0x06 then pad with zeros and set last of rate |= 0x80
    uint8_t block[72];
#pragma unroll
    for (int i = 0; i < 72; ++i) block[i] = 0;
#pragma unroll
    for (int i = 0; i < 64; ++i) block[i] = in_be64[i];
    block[64] = 0x06;
    block[71] ^= 0x80;
 
    // XOR into state lanes as little-endian 64-bit words
#pragma unroll
    for (int i = 0; i < 9; ++i) {
        s[i] ^= load64_le(&block[i * 8]);
    }
 
    // Permute
    keccak_f1600(s);
 
    // Squeeze 64 bytes (8 lanes)
    uint8_t out_le[64];
#pragma unroll
    for (int i = 0; i < 8; ++i) {
        store64_le(&out_le[i * 8], s[i]);
    }
 
    // The standard digest byte order expected by host libraries is the emitted byte stream.
    // We keep it as-is; host target/threshold bytes should be in the same orientation to XOR/compare.
#pragma unroll
    for (int i = 0; i < 64; ++i) {
        out_be64[i] = out_le[i];
    }
}
 
// Convert 8 LE limbs into 64 BE bytes (big-endian numeric representation)
__device__ __forceinline__ void le8_to_be64_bytes(const uint64_t le[8], uint8_t out[64]) {
#pragma unroll
    for (int i = 0; i < 8; ++i) {
        uint64_t limb = le[7 - i]; // most significant limb first
#pragma unroll
        for (int b = 0; b < 8; ++b) {
            out[i * 8 + (7 - b)] = (uint8_t)((limb >> (b * 8)) & 0xFF);
        }
    }
}
 
// Compare two 64-byte big-endian numbers: return true if a <= b
__device__ __forceinline__ bool be64_leq(const uint8_t a[64], const uint8_t b[64]) {
#pragma unroll
    for (int i = 0; i < 64; ++i) {
        if (a[i] != b[i]) {
            return a[i] < b[i];
        }
    }
    return true; // equal
}
 
// Kernel: G2 — device SHA3-512 + threshold compare + early-exit
//
// Notes:
// - Signature includes additional G2 parameters; host launcher must be updated to pass them.
// - Early-exit: a single global int flag claimed via atomicCAS; earliest winning thread writes result.
//
extern "C" __global__ void qpow_montgomery_g2_kernel(
    // Per-job constants (each 8 limbs, LE)
    const uint64_t* __restrict__ m,
    const uint64_t* __restrict__ n,
    const uint64_t  n0_inv,
    const uint64_t* __restrict__ r2,
    const uint64_t* __restrict__ m_hat,
 
    // Per-thread starting state (normal domain)
    const uint64_t* __restrict__ y0,            // length: num_threads * 8 limbs
 
    // G2-specific inputs/outputs
    const uint8_t*  __restrict__ target_be,     // 64 bytes
    const uint8_t*  __restrict__ threshold_be,  // 64 bytes
    int*            __restrict__ found_flag,    // 0 -> not found, 1 -> found
    uint32_t*       __restrict__ out_index,     // linear index (t * iters + j)
    uint8_t*        __restrict__ out_distance_be, // 64 bytes
 
    // Threading parameters
    const uint32_t num_threads,
    const uint32_t iters_per_thread
) {
    const uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_threads) {
        return;
    }
 
    // Quick early-exit check
    if (atomicAdd(found_flag, 0) != 0) {
        return;
    }
 
    // Local copies of constants
    uint64_t n_loc[8], r2_loc[8], mhat_loc[8];
#pragma unroll
    for (int i = 0; i < 8; ++i) {
        n_loc[i]    = n[i];
        r2_loc[i]   = r2[i];
        mhat_loc[i] = m_hat[i];
    }
 
    // Load this thread's y0 (normal domain) and move to Montgomery domain
    uint64_t y0_loc[8];
#pragma unroll
    for (int i = 0; i < 8; ++i) {
        y0_loc[i] = y0[tid * 8u + i];
    }
    uint64_t yhat[8];
    to_mont_512(y0_loc, r2_loc, n_loc, n0_inv, yhat);
 
    // Iterate and check threshold
    const uint32_t iters = iters_per_thread;
    for (uint32_t j = 0; j < iters; ++j) {
        // Respect early-exit
        if (atomicAdd(found_flag, 0) != 0) {
            return;
        }
 
        // y_hat = y_hat * m_hat
        uint64_t yhat_next[8];
        mont_mul_512(yhat, mhat_loc, n_loc, n0_inv, yhat_next);
#pragma unroll
        for (int i = 0; i < 8; ++i) {
            yhat[i] = yhat_next[i];
        }
 
        // y = from_mont(y_hat)
        uint64_t y_norm[8];
        from_mont_512(yhat, n_loc, n0_inv, y_norm);
 
        // y_be64 (64 bytes) from LE limbs
        uint8_t y_be[64];
        le8_to_be64_bytes(y_norm, y_be);
 
        // H = SHA3-512(y_be)
        uint8_t h_be[64];
        sha3_512_64bytes(y_be, h_be);
 
        // distance = target XOR H (byte-wise)
        uint8_t dist_be[64];
#pragma unroll
        for (int i = 0; i < 64; ++i) {
            dist_be[i] = target_be[i] ^ h_be[i];
        }
 
        // Compare distance <= threshold
        if (be64_leq(dist_be, threshold_be)) {
            // Try to claim the flag
            if (atomicCAS(found_flag, 0, 1) == 0) {
                // Write linear index for host to reconstruct nonce
                if (out_index) {
                    *out_index = tid * iters + j;
                }
                // Write distance
                if (out_distance_be) {
#pragma unroll
                    for (int i = 0; i < 64; ++i) {
                        out_distance_be[i] = dist_be[i];
                    }
                }
            }
            return; // early-exit after claiming
        }
    }
}
 
} // extern "C"