/**
 * Quantus External Miner - CUDA Kernel (G1 bring-up)
 *
 * This kernel provides a minimal, correctness-first pipeline for the GPU path:
 * - Implement 512-bit Montgomery multiplication (8×64-bit limbs) on device.
 * - Keep y in Montgomery domain during iteration and convert to normal domain before output.
 * - For bring-up (G1), the kernel writes normalized y values back to host memory.
 *   The host will compute SHA3-512(y_be64) and distances for parity validation.
 *
 * Notes:
 * - Limbs are little-endian: limb 0 is the least significant 64 bits.
 * - CIOS Montgomery reduction is used with 64×64→128 products via __umul64hi.
 * - This skeleton intentionally excludes early-exit and on-device SHA3; those are part of G2.
 *
 * Build:
 * - The engine-gpu-cuda crate provides a build.rs that compiles this .cu into PTX when the
 *   "cuda" feature is enabled, placing artifacts under $OUT_DIR and exposing ENGINE_GPU_CUDA_PTX_DIR.
 */

#include <stdint.h>
#include <hip/hip_runtime.h>

extern "C" {

// -------------------------------------------------------------------------------------------------
// Utilities: 64×64→128 multiply (lo, hi), add-with-carry helpers, compare/subtract
// -------------------------------------------------------------------------------------------------

__device__ __forceinline__ void mul64wide(uint64_t a, uint64_t b, uint64_t &lo, uint64_t &hi) {
    lo = a * b;
    hi = __umul64hi(a, b);
}

// sum := x + y, carry_out returns 0 or 1
__device__ __forceinline__ uint64_t add64_carry(uint64_t x, uint64_t y, uint64_t &carry_out) {
    uint64_t s = x + y;
    carry_out = (s < x) ? 1ull : 0ull;
    return s;
}

// sum := x + y + carry_in, carry_out returns 0 or 1
__device__ __forceinline__ uint64_t add64_2carry(uint64_t x, uint64_t y, uint64_t carry_in, uint64_t &carry_out) {
    uint64_t s1 = x + y;
    uint64_t c1 = (s1 < x) ? 1ull : 0ull;
    uint64_t s2 = s1 + carry_in;
    uint64_t c2 = (s2 < s1) ? 1ull : 0ull;
    carry_out = c1 + c2;
    return s2;
}

// return true if a (LE limbs) >= b (LE limbs), by numeric value
__device__ __forceinline__ bool ge_le_8(const uint64_t a[8], const uint64_t b[8]) {
    // Compare from most significant limb to least
    for (int i = 7; i >= 0; --i) {
        if (a[i] != b[i]) {
            return a[i] > b[i];
        }
    }
    return true; // equal
}

// a := a - b (LE limbs)
__device__ __forceinline__ void sub_le_in_place_8(uint64_t a[8], const uint64_t b[8]) {
    uint64_t borrow = 0;
    for (int i = 0; i < 8; ++i) {
        uint64_t bi = b[i];
        uint64_t ai = a[i];
        uint64_t tmp = ai - bi - borrow;
        // borrow occurs if ai < (bi + borrow)
        uint64_t needed = (ai < bi) || (borrow && ai == bi) ? 1ull : 0ull;
        a[i] = tmp;
        borrow = needed;
    }
}

// Convert 8 LE limbs into 64 BE bytes into out[64] (for host hashing later, if needed)
// Not used inside the kernel (G1 writes limbs), but kept here for reference.
// __device__ __forceinline__ void le8_to_be64(const uint64_t le[8], uint8_t out[64]) {
//     for (int i = 0; i < 8; ++i) {
//         uint64_t limb = le[7 - i]; // most significant limb first
//         for (int b = 0; b < 8; ++b) {
//             out[i * 8 + (7 - b)] = (uint8_t)((limb >> (b * 8)) & 0xFF);
//         }
//     }
// }

// -------------------------------------------------------------------------------------------------
// Montgomery arithmetic (CIOS) for 512-bit numbers (8×64-bit limbs), little-endian.
// -------------------------------------------------------------------------------------------------

// out <- (a * b * R^{-1}) mod n
__device__ __forceinline__ void mont_mul_512(
    const uint64_t a[8],
    const uint64_t b[8],
    const uint64_t n[8],
    const uint64_t n0_inv,
    uint64_t out[8]
) {
    // 9-limb accumulator (LE); accumulates 128-bit intermediates via split-add with carries
    uint64_t acc[9];
#pragma unroll
    for (int k = 0; k < 9; ++k) acc[k] = 0ull;

    for (int i = 0; i < 8; ++i) {
        // acc += a[i] * b
        uint64_t ai = a[i];
        uint64_t carry = 0ull;
#pragma unroll
        for (int j = 0; j < 8; ++j) {
            uint64_t lo, hi;
            mul64wide(ai, b[j], lo, hi);

            // acc[j] += lo + carry, propagate carry to hi
            uint64_t c0, c1;
            uint64_t s0 = add64_carry(acc[j], lo, c0);
            uint64_t s1 = add64_carry(s0, carry, c1);
            acc[j] = s1;
            // new carry = hi + c0 + c1
            carry = hi + c0 + c1;
        }
        // acc[8] += carry
        uint64_t c_acc8;
        acc[8] = add64_carry(acc[8], carry, c_acc8);
        // c_acc8 overflow beyond 9th limb is discarded (by design in CIOS with next steps)

        // m = (acc[0] * n0_inv) mod 2^64
        uint64_t m = (uint64_t)(acc[0] * n0_inv);

        // acc += m * n
        uint64_t carry2 = 0ull;
#pragma unroll
        for (int j = 0; j < 8; ++j) {
            uint64_t lo2, hi2;
            mul64wide(m, n[j], lo2, hi2);

            uint64_t c0, c1;
            uint64_t s0 = add64_carry(acc[j], lo2, c0);
            uint64_t s1 = add64_carry(s0, carry2, c1);
            acc[j] = s1;
            carry2 = hi2 + c0 + c1;
        }
        // acc[8] += carry2
        uint64_t c_acc8_b;
        acc[8] = add64_carry(acc[8], carry2, c_acc8_b);

        // Shift acc right by one limb (drop acc[0])
#pragma unroll
        for (int j = 0; j < 8; ++j) {
            acc[j] = acc[j + 1];
        }
        acc[8] = 0ull;
    }

    // Conditional subtract: if acc >= n, subtract n
    if (ge_le_8(acc, n)) {
        sub_le_in_place_8(acc, n);
    }

#pragma unroll
    for (int i = 0; i < 8; ++i) {
        out[i] = acc[i];
    }
}

// to_mont(x) = x * R^2 mod n
__device__ __forceinline__ void to_mont_512(
    const uint64_t x[8],
    const uint64_t r2[8],
    const uint64_t n[8],
    const uint64_t n0_inv,
    uint64_t out[8]
) {
    mont_mul_512(x, r2, n, n0_inv, out);
}

// from_mont(x̂) = x̂ * 1 mod n
__device__ __forceinline__ void from_mont_512(
    const uint64_t xhat[8],
    const uint64_t n[8],
    const uint64_t n0_inv,
    uint64_t out[8]
) {
    // Multiply by 1 (Montgomery): one = [1,0,..,0]
    uint64_t one[8];
#pragma unroll
    for (int i = 0; i < 8; ++i) one[i] = 0ull;
    one[0] = 1ull;
    mont_mul_512(xhat, one, n, n0_inv, out);
}

// -------------------------------------------------------------------------------------------------
// Kernel: G1 bring-up
//
// Each thread:
//  - Loads y0 (normal domain) for that thread.
//  - Computes y_hat0 = to_mont(y0) on device.
//  - Iterates iters_per_thread times:
//      y_hat = mont_mul(y_hat, m_hat)
//      y = from_mont(y_hat)
//      Writes y (LE limbs) to y_out at [thread_offset + iter]
// -------------------------------------------------------------------------------------------------

extern "C" __global__ void qpow_montgomery_g1_kernel(
    // Per-job constants (each 8 limbs, LE)
    const uint64_t* __restrict__ m,        // not used in G1 directly (we pass m_hat)
    const uint64_t* __restrict__ n,
    const uint64_t  n0_inv,
    const uint64_t* __restrict__ r2,
    const uint64_t* __restrict__ m_hat,

    // Per-thread starting state (normal domain)
    const uint64_t* __restrict__ y0,       // length: num_threads * 8 limbs

    // Output buffer for normalized y (for host SHA3 in G1)
    uint64_t* __restrict__ y_out,          // length: num_threads * iters_per_thread * 8 limbs

    // Threading parameters
    const uint32_t num_threads,
    const uint32_t iters_per_thread
) {
    const uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_threads) {
        return;
    }

    // Local copies of constants (consider placing in __constant__ memory for G2+)
    uint64_t n_loc[8], r2_loc[8], mhat_loc[8];
#pragma unroll
    for (int i = 0; i < 8; ++i) {
        n_loc[i]    = n[i];
        r2_loc[i]   = r2[i];
        mhat_loc[i] = m_hat[i];
    }

    // Load this thread's y0 (normal domain)
    uint64_t y0_loc[8];
#pragma unroll
    for (int i = 0; i < 8; ++i) {
        y0_loc[i] = y0[tid * 8u + i];
    }

    // Transform to Montgomery domain
    uint64_t yhat[8];
    to_mont_512(y0_loc, r2_loc, n_loc, n0_inv, yhat);

    // Iterate and emit normalized y per step
    // Output stride per thread: iters_per_thread * 8 limbs
    uint64_t* out_base = y_out + (static_cast<size_t>(tid) * static_cast<size_t>(iters_per_thread) * 8ull);

    for (uint32_t iter = 0; iter < iters_per_thread; ++iter) {
        // y_hat = y_hat * m_hat
        uint64_t yhat_next[8];
        mont_mul_512(yhat, mhat_loc, n_loc, n0_inv, yhat_next);

#pragma unroll
        for (int i = 0; i < 8; ++i) {
            yhat[i] = yhat_next[i];
        }

        // y = from_mont(y_hat)
        uint64_t y_norm[8];
        from_mont_512(yhat, n_loc, n0_inv, y_norm);

        // Store normalized y (LE limbs) for host SHA3 and distance validation
#pragma unroll
        for (int i = 0; i < 8; ++i) {
            out_base[iter * 8u + i] = y_norm[i];
        }
    }
}

// -------------------------------------------------------------------------------------------------
// Host-callable launcher wrapper (optional; typically loaded via PTX and launched from Rust)
// -------------------------------------------------------------------------------------------------

// The Rust host side will load this kernel from PTX and launch it using `cust`/`rustacuda`.
// Example signature in Rust (pseudo):
//
// launch!(module.qpow_montgomery_g1_kernel<<<grid, block, 0, stream>>>(
//     d_m.as_device_ptr(),
//     d_n.as_device_ptr(),
//     n0_inv,
//     d_r2.as_device_ptr(),
//     d_mhat.as_device_ptr(),
//     d_y0.as_device_ptr(),
//     d_y_out.as_device_ptr(),
//     num_threads,
//     iters_per_thread
// ))?;
//
// Note: For G1, the host will compute SHA3-512(y) and distances on the CPU,
// validating correctness against cpu-fast/cpu-montgomery on small ranges.
//

// -------------------------------------------------------------------------------------------------
// G2 additions: device SHA3-512, threshold compare, and early-exit
// -------------------------------------------------------------------------------------------------

// 64-bit rotate-left with defined behavior for all n
__device__ __forceinline__ uint64_t rotl64(uint64_t x, unsigned int n) {
n &= 63u;
return (x << n) | (x >> ((64u - n) & 63u));
}

// Keccak-f[1600] round constants
__device__ __constant__ uint64_t KECCAK_RC[24] = {
0x0000000000000001ULL, 0x0000000000008082ULL,
0x800000000000808aULL, 0x8000000080008000ULL,
0x000000000000808bULL, 0x0000000080000001ULL,
0x8000000080008081ULL, 0x8000000000008009ULL,
0x000000000000008aULL, 0x0000000000000088ULL,
0x0000000080008009ULL, 0x000000008000000aULL,
0x000000008000808bULL, 0x800000000000008bULL,
0x8000000000008089ULL, 0x8000000000008003ULL,
0x8000000000008002ULL, 0x8000000000000080ULL,
0x000000000000800aULL, 0x800000008000000aULL,
0x8000000080008081ULL, 0x8000000000008080ULL,
0x0000000080000001ULL, 0x8000000080008008ULL
};

// Optional per-job constants in constant memory (host may set; kernel remains compatible)
// If C_CONSTS_READY == 1, G2 kernel will prefer these over parameter pointers.
__device__ __constant__ uint64_t C_N[8];
__device__ __constant__ uint64_t C_R2[8];
__device__ __constant__ uint64_t C_MHAT[8];
__device__ __constant__ uint64_t C_N0_INV;
__device__ __constant__ int      C_CONSTS_READY;
// Optional constant-memory target/threshold for device compare
__device__ __constant__ uint64_t C_TARGET[8];
__device__ __constant__ uint64_t C_THRESH[8];
// Optional sampler controls/output (host may read these symbols when enabled)
__device__ __constant__ int      C_SAMPLER_ENABLE;
__device__ uint8_t               C_SAMPLER_Y_BE[64];
__device__ uint8_t               C_SAMPLER_H_BE[64];
__device__ uint8_t               C_SAMPLER_TARGET_BE[64];
__device__ uint8_t               C_SAMPLER_THRESH_BE[64];
__device__ uint32_t              C_SAMPLER_INDEX;
__device__ uint32_t              C_SAMPLER_DECISION;
__device__ uint32_t              C_WIN_TID;
__device__ uint32_t              C_WIN_J;

// Load/store helpers (little- and big-endian)
__device__ __forceinline__ uint64_t load64_le(const uint8_t* p) {
return ((uint64_t)p[0])       |
       ((uint64_t)p[1] << 8)  |
       ((uint64_t)p[2] << 16) |
       ((uint64_t)p[3] << 24) |
       ((uint64_t)p[4] << 32) |
       ((uint64_t)p[5] << 40) |
       ((uint64_t)p[6] << 48) |
       ((uint64_t)p[7] << 56);
}
__device__ __forceinline__ void store64_le(uint8_t* p, uint64_t v) {
p[0] = (uint8_t)(v);
p[1] = (uint8_t)(v >> 8);
p[2] = (uint8_t)(v >> 16);
p[3] = (uint8_t)(v >> 24);
p[4] = (uint8_t)(v >> 32);
p[5] = (uint8_t)(v >> 40);
p[6] = (uint8_t)(v >> 48);
p[7] = (uint8_t)(v >> 56);
}
__device__ __forceinline__ void store64_be(uint8_t* p, uint64_t v) {
p[0] = (uint8_t)(v >> 56);
p[1] = (uint8_t)(v >> 48);
p[2] = (uint8_t)(v >> 40);
p[3] = (uint8_t)(v >> 32);
p[4] = (uint8_t)(v >> 24);
p[5] = (uint8_t)(v >> 16);
p[6] = (uint8_t)(v >> 8);
p[7] = (uint8_t)(v);
}

// Convert 8 LE limbs into 64 BE bytes (big-endian numeric representation)
__device__ __forceinline__ void le8_to_be64_bytes(const uint64_t le[8], uint8_t out[64]) {
#pragma unroll
for (int i = 0; i < 8; ++i) {
    uint64_t limb = le[7 - i]; // most significant limb first
#pragma unroll
    for (int b = 0; b < 8; ++b) {
        out[i * 8 + (7 - b)] = (uint8_t)((limb >> (b * 8)) & 0xFF);
    }
}
}

// Compare two 64-byte big-endian numbers: return true if a <= b
__device__ __forceinline__ bool be64_leq(const uint8_t a[64], const uint8_t b[64]) {
#pragma unroll
for (int i = 0; i < 64; ++i) {
    if (a[i] != b[i]) {
        return a[i] < b[i];
    }
}
return true; // equal
}

// Keccak-f[1600] permutation (unrolled mapping)
__device__ __forceinline__ void keccak_f1600(uint64_t s[25]) {
#pragma unroll
for (int round = 0; round < 24; ++round) {
    // ---- Theta ---------------------------------------------------------
    uint64_t Aba=s[0],  Aga=s[5],  Aka=s[10],  Ama=s[15],  Asa=s[20];
    uint64_t Abe=s[1],  Age=s[6],  Ake=s[11],  Ame=s[16],  Ase=s[21];
    uint64_t Abi=s[2],  Agi=s[7],  Aki=s[12],  Ami=s[17],  Asi=s[22];
    uint64_t Abo=s[3],  Ago=s[8],  Ako=s[13],  Amo=s[18],  Aso=s[23];
    uint64_t Abu=s[4],  Agu=s[9],  Aku=s[14],  Amu=s[19],  Asu=s[24];

    uint64_t Ca = Aba ^ Aga ^ Aka ^ Ama ^ Asa;
    uint64_t Ce = Abe ^ Age ^ Ake ^ Ame ^ Ase;
    uint64_t Ci = Abi ^ Agi ^ Aki ^ Ami ^ Asi;
    uint64_t Co = Abo ^ Ago ^ Ako ^ Amo ^ Aso;
    uint64_t Cu = Abu ^ Agu ^ Aku ^ Amu ^ Asu;

    uint64_t Da = rotl64(Ce, 1) ^ Cu;
    uint64_t De = rotl64(Ci, 1) ^ Ca;
    uint64_t Di = rotl64(Co, 1) ^ Ce;
    uint64_t Do = rotl64(Cu, 1) ^ Ci;
    uint64_t Du = rotl64(Ca, 1) ^ Co;

    Aba ^= Da; Abe ^= De; Abi ^= Di; Abo ^= Do; Abu ^= Du;
    Aga ^= Da; Age ^= De; Agi ^= Di; Ago ^= Do; Agu ^= Du;
    Aka ^= Da; Ake ^= De; Aki ^= Di; Ako ^= Do; Aku ^= Du;
    Ama ^= Da; Ame ^= De; Ami ^= Di; Amo ^= Do; Amu ^= Du;
    Asa ^= Da; Ase ^= De; Asi ^= Di; Aso ^= Do; Asu ^= Du;

    // ---- Rho + Pi ------------------------------------------------------
    uint64_t Bba = Aba;
    uint64_t Bbe = rotl64(Age, 44);
    uint64_t Bbi = rotl64(Aki, 43);
    uint64_t Bbo = rotl64(Amo, 21);
    uint64_t Bbu = rotl64(Asu, 14);

    uint64_t Bga = rotl64(Abo, 28);
    uint64_t Bge = rotl64(Agu, 20);
    uint64_t Bgi = rotl64(Aka, 3);
    uint64_t Bgo = rotl64(Ame, 45);
    uint64_t Bgu = rotl64(Asi, 61);

    uint64_t Bka = rotl64(Abe, 1);
    uint64_t Bke = rotl64(Agi, 6);
    uint64_t Bki = rotl64(Ako, 25);
    uint64_t Bko = rotl64(Amu, 8);
    uint64_t Bku = rotl64(Asa, 18);

    uint64_t Bma = rotl64(Abu, 27);
    uint64_t Bme = rotl64(Aga, 36);
    uint64_t Bmi = rotl64(Ake, 10);
    uint64_t Bmo = rotl64(Ami, 15);
    uint64_t Bmu = rotl64(Aso, 56);

    uint64_t Bsa = rotl64(Abi, 62);
    uint64_t Bse = rotl64(Ago, 55);
    uint64_t Bsi = rotl64(Aku, 39);
    uint64_t Bso = rotl64(Ama, 41);
    uint64_t Bsu = rotl64(Ase, 2);

    // ---- Chi -----------------------------------------------------------
    Aba = Bba ^ ((~Bbe) & Bbi);
    Abe = Bbe ^ ((~Bbi) & Bbo);
    Abi = Bbi ^ ((~Bbo) & Bbu);
    Abo = Bbo ^ ((~Bbu) & Bba);
    Abu = Bbu ^ ((~Bba) & Bbe);

    Aga = Bga ^ ((~Bge) & Bgi);
    Age = Bge ^ ((~Bgi) & Bgo);
    Agi = Bgi ^ ((~Bgo) & Bgu);
    Ago = Bgo ^ ((~Bgu) & Bga);
    Agu = Bgu ^ ((~Bga) & Bge);

    Aka = Bka ^ ((~Bke) & Bki);
    Ake = Bke ^ ((~Bki) & Bko);
    Aki = Bki ^ ((~Bko) & Bku);
    Ako = Bko ^ ((~Bku) & Bka);
    Aku = Bku ^ ((~Bka) & Bke);

    Ama = Bma ^ ((~Bme) & Bmi);
    Ame = Bme ^ ((~Bmi) & Bmo);
    Ami = Bmi ^ ((~Bmo) & Bmu);
    Amo = Bmo ^ ((~Bmu) & Bma);
    Amu = Bmu ^ ((~Bma) & Bme);

    Asa = Bsa ^ ((~Bse) & Bsi);
    Ase = Bse ^ ((~Bsi) & Bso);
    Asi = Bsi ^ ((~Bso) & Bsu);
    Aso = Bso ^ ((~Bsu) & Bsa);
    Asu = Bsu ^ ((~Bsa) & Bse);

    // ---- Iota ----------------------------------------------------------
    Aba ^= KECCAK_RC[round];

    // Store back
    s[0]=Aba;  s[5]=Aga;  s[10]=Aka;  s[15]=Ama;  s[20]=Asa;
    s[1]=Abe;  s[6]=Age;  s[11]=Ake;  s[16]=Ame;  s[21]=Ase;
    s[2]=Abi;  s[7]=Agi;  s[12]=Aki;  s[17]=Ami;  s[22]=Asi;
    s[3]=Abo;  s[8]=Ago;  s[13]=Ako;  s[18]=Amo;  s[23]=Aso;
    s[4]=Abu;  s[9]=Agu;  s[14]=Aku;  s[19]=Amu;  s[24]=Asu;
}
}

// Device SHA3-512 for a single 64-byte message; writes lane-LE bytes to out_le64
// Note: Input is treated as raw message bytes. We absorb them directly into the Keccak rate
// as little-endian 64-bit lanes to mirror the host sha3 crate semantics.
__device__ __forceinline__ void sha3_512_64bytes_le(const uint8_t in_msg_bytes[64], uint8_t out_le64[64]) {
// Initialize state to zero
uint64_t s[25];
#pragma unroll
for (int i = 0; i < 25; ++i) s[i] = 0ull;

// Absorb (rate = 72 bytes). Message is 64 bytes: append 0x06 then pad with zeros and set last of rate |= 0x80
uint8_t block[72];
#pragma unroll
for (int i = 0; i < 72; ++i) block[i] = 0;
#pragma unroll
for (int i = 0; i < 64; ++i) block[i] = in_msg_bytes[i];
block[64] = 0x06;
block[71] ^= 0x80;

// XOR into state lanes as little-endian 64-bit words
#pragma unroll
for (int i = 0; i < 9; ++i) {
    s[i] ^= load64_le(&block[i * 8]);
}

// Permute
keccak_f1600(s);

// Squeeze 64 bytes (8 lanes) into little-endian lane bytes
#pragma unroll
for (int i = 0; i < 8; ++i) {
    store64_le(&out_le64[i * 8], s[i]);
}
}

// Kernel: G2 — device SHA3-512 + threshold compare + early-exit
extern "C" __global__ void qpow_montgomery_g2_kernel(
// Per-job constants (each 8 limbs, LE)
const uint64_t* __restrict__ m,
const uint64_t* __restrict__ n,
const uint64_t  n0_inv,
const uint64_t* __restrict__ r2,
const uint64_t* __restrict__ m_hat,

// Per-thread starting state (normal domain)
const uint64_t* __restrict__ y0,            // length: num_threads * 8 limbs

// G2-specific inputs/outputs
const uint8_t*  __restrict__ target_be,     // 64 bytes
const uint8_t*  __restrict__ threshold_be,  // 64 bytes
int*            __restrict__ found_flag,    // 0 -> not found, 1 -> found
uint32_t*       __restrict__ out_index,     // linear index (t * iters + j)
uint8_t*        __restrict__ out_distance_be, // 64 bytes
// Debug output buffers (optional; host may pass nullptrs)
uint8_t*        __restrict__ out_dbg_y_be,    // 64 bytes (optional)
uint8_t*        __restrict__ out_dbg_h_be,    // 64 bytes (optional)

// Threading parameters
const uint32_t num_threads,
const uint32_t iters_per_thread
) {
const uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= num_threads) {
    return;
}

// Quick early-exit check
if (atomicAdd(found_flag, 0) != 0) {
    return;
}

// Local copies of constants (prefer __constant__ if available)
uint64_t n_loc[8], r2_loc[8], mhat_loc[8];
if (C_CONSTS_READY) {
#pragma unroll
    for (int i = 0; i < 8; ++i) {
        n_loc[i]    = C_N[i];
        r2_loc[i]   = C_R2[i];
        mhat_loc[i] = C_MHAT[i];
    }
} else {
#pragma unroll
    for (int i = 0; i < 8; ++i) {
        n_loc[i]    = n[i];
        r2_loc[i]   = r2[i];
        mhat_loc[i] = m_hat[i];
    }
}

// Load this thread's y0 (normal domain) and move to Montgomery domain
uint64_t y0_loc[8];
#pragma unroll
for (int i = 0; i < 8; ++i) {
    y0_loc[i] = y0[tid * 8u + i];
}
const uint64_t n0i = C_CONSTS_READY ? C_N0_INV : n0_inv;
uint64_t yhat[8];
to_mont_512(y0_loc, r2_loc, n_loc, n0i, yhat);

// Prepare target/threshold big-endian bytes (numeric)
uint8_t target_be_bytes[64], thresh_be_bytes[64];
if (C_CONSTS_READY) {
#pragma unroll
    for (int i = 0; i < 8; ++i) {
        store64_be(&target_be_bytes[i * 8], C_TARGET[i]);
        store64_be(&thresh_be_bytes[i * 8], C_THRESH[i]);
    }
} else {
#pragma unroll
    for (int i = 0; i < 64; ++i) {
        target_be_bytes[i] = target_be[i];
        thresh_be_bytes[i] = threshold_be[i];
    }
}

// Iterate and check threshold
const uint32_t iters = iters_per_thread;
for (uint32_t j = 0; j < iters; ++j) {
    // Respect early-exit
    if (atomicAdd(found_flag, 0) != 0) {
        return;
    }

    // y_hat = y_hat * m_hat
    uint64_t yhat_next[8];
    mont_mul_512(yhat, mhat_loc, n_loc, n0i, yhat_next);
#pragma unroll
    for (int i = 0; i < 8; ++i) {
        yhat[i] = yhat_next[i];
    }

    // y = from_mont(y_hat)
    uint64_t y_norm[8];
    from_mont_512(yhat, n_loc, n0i, y_norm);

    // y_be64 (64 bytes) from LE limbs
    uint8_t y_be[64];
    le8_to_be64_bytes(y_norm, y_be);

    // H = SHA3-512(y_be) -> produce lane-LE bytes
    uint8_t h_le[64];
    sha3_512_64bytes_le(y_be, h_le);

    // Convert digest to big-endian numeric bytes (preserve lane order; convert each lane LE->BE in-place)
        uint8_t digest_be[64];
        #pragma unroll
        for (int i = 0; i < 8; ++i) {
            uint64_t w = load64_le(&h_le[i * 8]);
            store64_be(&digest_be[i * 8], w);
        }

    // distance = target_be XOR digest_be (bytewise, big-endian order)
    uint8_t dist_be[64];
#pragma unroll
    for (int i = 0; i < 64; ++i) {
        dist_be[i] = target_be_bytes[i] ^ h_le[i];
    }

    // Compare distance <= threshold (lexicographic on big-endian bytes)
    bool decision = be64_leq(dist_be, thresh_be_bytes);

    // Optional sampler (first thread/iter): capture y/H/target/thresh for parity
    if (C_SAMPLER_ENABLE && tid == 0 && j == 0) {
#pragma unroll
        for (int i = 0; i < 64; ++i) {
            C_SAMPLER_Y_BE[i]       = y_be[i];
            C_SAMPLER_H_BE[i]       = h_le[i];
            C_SAMPLER_TARGET_BE[i]  = target_be_bytes[i];
            C_SAMPLER_THRESH_BE[i]  = thresh_be_bytes[i];
        }
        C_SAMPLER_INDEX = tid * iters + j;
        C_SAMPLER_DECISION = decision ? 1u : 0u;
    }

    if (decision) {
        // Try to claim the flag
        if (atomicCAS(found_flag, 0, 1) == 0) {
            // Write linear index for host to reconstruct nonce
            if (out_index) {
                *out_index = tid * iters + j;
            }
            // Record winner thread and iteration for host-side nonce reconstruction
            C_WIN_TID = tid;
            C_WIN_J = j;
            // Write distance and debug buffers (if provided)
            if (out_distance_be) {
#pragma unroll
                for (int i = 0; i < 64; ++i) {
                    out_distance_be[i] = dist_be[i];
                }
            }
            if (out_dbg_y_be) {
#pragma unroll
                for (int i = 0; i < 64; ++i) {
                    out_dbg_y_be[i] = y_be[i];
                }
            }
            if (out_dbg_h_be) {
#pragma unroll
                for (int i = 0; i < 64; ++i) {
                    out_dbg_h_be[i] = h_le[i];
                }
            }
        }
        return; // early-exit after claiming (or if already claimed)
    }
}
}

} // extern "C"
